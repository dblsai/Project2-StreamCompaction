#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include "ParallelReductioin.h"
#include "stdio.h"

// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200) 
// # error printf is only supported on devices of compute capability 2.0 and higher, please compile with -arch=sm_20 or higher 
// #endif

hipEvent_t start, stop;
float timeDuration;
int * OUTd, * INd, * TMPd;
int LEN;
int * SUMS;   //auxilary memory to store the sums for each block

struct is_odd
  {
    __host__ __device__
    bool operator()(const int x)
    {
      return (x % 2) == 1;
    }
  };

__device__ int predictCondition(int a){
	if( a % 2 == 0 ){   //the condition here
		return 0;
	}
	else{
		return 1;
	}
}

//parallelize this using a series of kernel calls
//NOT allowed to use shared memory
__global__ void globalReduction(int* INd, int * OUTd, int * TMPd, int LENd){
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	int p2 = 0, p1 = 1, tmp; 
	//predict
	if(index<LENd){
		TMPd[index] = (threadIdx.x==0 && blockIdx.x ==0)? 0:predictCondition (INd[index-1]);
	}
	__syncthreads();

	//scan
	int round = (int)ceil( log2( (double)LENd ) );
	for(int d =1; d<=round; d++){
		//swap left and right buffer
		tmp = p2; p2 = 1-tmp;p1 = tmp;  
		int num = (int)std::pow((float)2,(float)(d-1));
		if(index<LENd){
			if(index>=num){
				
				TMPd[p2*LENd + index] = TMPd[ p1*LENd + index ] + TMPd[ p1*LENd + index - num];
			}
			else{
				TMPd[p2*LENd + index] = TMPd[ p1*LENd + index ];
			}
		}
		__syncthreads();
	}

	//scatter
	//OUTd[index] = TMPd[p2*LENd + index];
	if( index < LENd && predictCondition (INd[index]) == 1 ){
		//OUTd[index] = TMPd[p2*LENd + index];
		OUTd[TMPd[p2*LENd + index]] = INd[index];
	}
	
}

//works on a single block
//shared memory
__global__ void sharedReductionSingle(int *INd, int * OUTd, int LENd){
	extern __shared__ int sharedOUT[];  // allocated on invocation, double buffer  
	int tx = threadIdx.x;

	int p2 = 0, p1 = 1, tmp; 
	//sharedOUT [ p2*LENd + tx ] = OUTd[tx];  
	//sharedOUT [ p2*LENd + tx ] = (tx > 0) ? INd[tx-1] : 0;   //shift to right, store in left buffer
	if(tx<LENd){
		sharedOUT [ p2*LENd + tx ] = (tx>0) ? predictCondition(INd[tx-1]) : 0;
	}
	__syncthreads(); 

	//scan
	int round = ceil( log2( (double)LENd ) );
	for(int d =1; d<=round; d++){
		//swap left and right buffer
		tmp = p2; p2 = 1-tmp; p1 = tmp;  
		int num = (int)std::pow((float)2,(float)(d-1));
		if(tx<LENd){
			if(tx>=num){
				sharedOUT[p2*LENd + tx] = sharedOUT[ p1*LENd + tx ] + sharedOUT[ p1*LENd + tx - num];
			}
			else{
				sharedOUT[p2*LENd + tx] = sharedOUT[ p1*LENd + tx ];
			}
		}
		__syncthreads();
	}

	//scatter
	if(tx<LENd && predictCondition (INd[tx]) == 1){
		OUTd[sharedOUT[ p2*LENd + tx]] = INd[tx];
	}
}


/*__global__ void prescan(int *g_odata, int *g_idata, int n)  
{  
	extern __shared__ float temp[];  // allocated on invocation  
	int thid = threadIdx.x;  
	int offset = 1;  

	temp[2*thid] = g_idata[2*thid]; // load input into shared memory  
	temp[2*thid+1] = g_idata[2*thid+1];

	 // build sum in place up the tree	
	for (int d = n>>1; d > 0; d >>= 1){  
		__syncthreads();  
		if (thid < d)  {  
			int ai = offset*(2*thid+1)-1;  
			int bi = offset*(2*thid+2)-1;  
			temp[bi] += temp[ai];  
		}  
		offset *= 2;  
	}
	if (thid == 0) { temp[n - 1] = 0; } // clear the last element  
                 
	// traverse down tree & build scan 
	for (int d = 1; d < n; d *= 2){  
		 offset >>= 1;  
		 __syncthreads();  
		 if (thid < d){  
			int ai = offset*(2*thid+1)-1;  
			int bi = offset*(2*thid+2)-1;  
			float t = temp[ai];  
			temp[ai] = temp[bi];  
			temp[bi] += t;   
		  }  
	}  
	 __syncthreads();  

	g_odata[2*thid] = temp[2*thid]; // write results to device memory  
	g_odata[2*thid+1] = temp[2*thid+1];  
 	
}  */

//fit for length bigger that a block capacity
//shared memory
__global__ void sharedReductionMultiple(int * INd,  int * OUTd, int LENd, int * SUMS){

	extern __shared__ int sharedOUT2[];
	//int half = (int)ceil(((float)BlockSize)/((float)2));
	int half = blockDim.x;
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int index = tx + bx*half;   //index to access global memory
	int p2 = 0, p1 = 1, tmp; 
	 
	//establish shared memory
	if(index < LENd ){
		sharedOUT2 [ p2*half+ tx ] = (tx==0 && bx==0)? 0:predictCondition (INd[index - 1]);
	}
	__syncthreads();

	//scan for a block
	int round = ceil( log2( (double)half) );
	for(int d =1; d<=round; d++){
		//swap left and right buffer
		tmp = p2; p2 = 1-tmp; p1 = tmp;  
		int num = (int)std::pow((float)2,(float)(d-1));
		if( tx<half ){
			if(tx>=num){
				sharedOUT2[ p2*half + tx] = sharedOUT2[ p1*half + tx ] + sharedOUT2[ p1*half + tx - num];
			}
			else{
				sharedOUT2[ p2*half + tx] = sharedOUT2[ p1*half + tx ];
			}
		}
		__syncthreads();
	}

	//add auxilary sum
	SUMS[bx] = sharedOUT2[ p2*half + half-1] ;
	if(bx>0){
		for(int k =0; k < bx; k++){
			sharedOUT2[ p2*half + tx] += SUMS[k];
			sharedOUT2[ p1*half + tx] += SUMS[k];
		}
	}
	__syncthreads();
	//scatter
	if( index < LENd && predictCondition (INd[index]) ==1 ){
		OUTd[ sharedOUT2[ p2*half + tx ] ] = INd[ index ];
	}
	
}

void ParallelReduction(int * IN, int * OUT, int Len, int OPERATION){
	LEN = Len;

	//load OUT device memory
	const int size = LEN*sizeof(int);
	hipMalloc((void**)&OUTd, size);
	hipMemcpy(OUTd, OUT, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&INd, size);
	hipMemcpy(INd, IN, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&TMPd, 2*size);   //double buffer array

	//cuda timer event
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//kernel invocation
	switch(OPERATION){

	case GLOBAL_MEM:     //parallel reduction in global memory
		{
			hipEventRecord( start, 0 );
			dim3 dimGrid((int)ceil((float)LEN/(float)BLOCK_SIZE),1);    //blocks per grid
			dim3 dimBlock(BLOCK_SIZE,1);    //threads per block
			globalReduction<<<dimGrid, dimBlock>>>(INd, OUTd, TMPd, LEN);
			hipEventRecord( stop, 0 );
			hipEventSynchronize( stop );
			hipEventElapsedTime( &timeDuration, start, stop );
			printf("\n\n\n*****************************************************\n");
			printf("Time Taken for GPU Global Memory : %f ms\n",timeDuration);
			printf("*****************************************************\n");
			break;
		}
	case SHARED_MEM_1:   //parallel reduction on shared memory with single block
		{
			dim3 dimGrid1(1,1);    //blocks per grid
			dim3 dimBlock1(LEN,1);    //threads per block

			hipEventRecord( start, 0 );
			sharedReductionSingle<<<dimGrid1, dimBlock1, 2*size>>>(INd, OUTd, LEN);
			hipEventRecord( stop, 0 );
			hipEventSynchronize( stop );
			hipEventElapsedTime( &timeDuration, start, stop );
			printf("\n\n\n*****************************************************\n");
			printf("Time Taken for GPU Shared Memory Single: %f ms\n",timeDuration);
			printf("*****************************************************\n");
			break;
		}
	case SHARED_MEM_N:
		{
			int blocknum = 2* (int)ceil((float)LEN/(float)BLOCK_SIZE);
			dim3 dimGrid2(blocknum,1);    //blocks per grid, N/B
			dim3 dimBlock2((int)ceil((float)BLOCK_SIZE/(float)2),1);    //threads per block, B/2
			hipMalloc((void**)&SUMS, blocknum*sizeof(int));
			hipEventRecord( start, 0 );
			sharedReductionMultiple<<<dimGrid2, dimBlock2, BLOCK_SIZE*sizeof(int)>>>(INd, OUTd, LEN , SUMS);
			hipEventRecord( stop, 0 );
			hipEventSynchronize( stop );
			hipEventElapsedTime( &timeDuration, start, stop );
			printf("\n\n\n*****************************************************\n");
			printf("Time Taken for GPU Shared Memory Multiple: %f ms\n",timeDuration);
			printf("*****************************************************\n");
			break;
		}

		case THRUST:
		{
			hipEventRecord( start, 0 );
			thrust::copy_if(IN, IN + LEN, OUT, is_odd());
			hipEventRecord( stop, 0 );
			hipEventSynchronize( stop );
			hipEventElapsedTime( &timeDuration, start, stop );
			printf("\n\n\n*****************************************************\n");
			printf("Time Taken for Thrust Method: %f ms\n",timeDuration);
			printf("*****************************************************\n");
			break;
		}

	}

	//read OUT from device
	hipMemcpy(OUT, OUTd, size, hipMemcpyDeviceToHost);

	//free memory
	hipFree(OUTd);
	hipFree(INd);
	hipFree(SUMS);
	hipFree(TMPd);

	//destroy timer event
	hipEventDestroy( start );
	hipEventDestroy( stop );
}